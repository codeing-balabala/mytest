// #include <stdio.h>
// #include <iostream>
// // #include "book.h"

// // #include "foo.cuh"
// __device__ int add1(int a, int b) { return a + b; }

// __global__ void add(int a, int b, int *c) { *c = add1(a, b); }

// int main(void) {
//   int c;
//   int *dev_c;
//   std::cout << "Hello NVCC" << std::endl;
//   cudaMalloc((void **)&dev_c, sizeof(int));
//   add<<<1, 1>>>(2, 7, dev_c);
//   // add(1, 2, dev_c);
//   cudaMemcpy(&c, dev_c, sizeof(int), cudaMemcpyDeviceToHost);
//   printf("2 + 7 = %d\n", c);

//   cudaFree(dev_c);
//   return 0;
// }

#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

// 二：线程执行代码
__global__ void vector_add(float* vec1, float* vec2, float* vecres, int length,
                           int* idx) {
  int tid = threadIdx.y * blockDim.x +
            threadIdx.x;  // 使用了threadIdx.x, threadIdx.x, blockDim.x
  *idx = tid;
  if (tid < length) {
    vecres[tid] = vec1[tid] + vec2[tid];
  }
}

int main() {
  const int length = 16;                  // 数组长度为16
  float a[length], b[length], c[length];  // host中的数组
  for (int i = 0; i < length; i++) {      // 初始赋值
    a[i] = b[i] = i;
  }
  float *a_device, *b_device, *c_device;  // device中的数组
  int* idx;
  int tid;

  hipMalloc((void**)&a_device, length * sizeof(float));  // 分配内存
  hipMalloc((void**)&b_device, length * sizeof(float));
  hipMalloc((void**)&c_device, length * sizeof(float));
  hipMalloc((void**)&idx, sizeof(int));

  hipMemcpy(a_device, a, length * sizeof(float),
             hipMemcpyHostToDevice);  // 将host数组的值拷贝给device数组
  hipMemcpy(b_device, b, length * sizeof(float), hipMemcpyHostToDevice);

  // 一：参数配置
  dim3 grid(1, 1, 1), block(length, 2, 1);  // 设置参数
  vector_add<<<grid, block>>>(a_device, b_device, c_device, length,
                              idx);  // 启动kernel

  hipMemcpy(c, c_device, length * sizeof(float),
             hipMemcpyDeviceToHost);  // 将结果拷贝到host
  hipMemcpy(&tid, idx, sizeof(int),
             hipMemcpyDeviceToHost);  // 将结果拷贝到host

  cout << "tid:" << tid << " \n";

  for (int i = 0; i < length; i++) {  // 打印出来方便观察
    cout << c[i] << " ";
  }
  cout << endl;

  system("pause");
  return 0;
}
